#include "hip/hip_runtime.h"
/*
 * This code is released into the public domain.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
 * EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
 * MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
 * IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS IN THE SOFTWARE.
 */

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <ctime>
#include <cfloat>

#include <algorithm>
#include <chrono>
#include <iomanip>
#include <iostream>
#include <map>
#include <memory>
#include <random>
#include <sstream>
#include <string>
#include <vector>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

using namespace std;

//////////////////////////////////////////////////////////////////////////////
// Error handling
// Adapted from the CUDNN classification code 
// sample: https://developer.nvidia.com/cuDNN

#define FatalError(s) do {                                             \
    std::stringstream _where, _message;                                \
    _where << __FILE__ << ':' << __LINE__;                             \
    _message << std::string(s) + "\n" << __FILE__ << ':' << __LINE__;  \
    std::cerr << _message.str() << "\nAborting...\n";                  \
    hipDeviceReset();                                                 \
    exit(1);                                                           \
} while(0)

#define checkCUDNN(status) do {                                        \
    std::stringstream _error;                                          \
    if (status != HIPDNN_STATUS_SUCCESS) {                              \
      _error << "CUDNN failure: " << hipdnnGetErrorString(status);      \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

#define checkCudaErrors(status) do {                                   \
    std::stringstream _error;                                          \
    if (status != 0) {                                                 \
      _error << "Cuda failure: " << status;                            \
			_error << hipGetErrorString(status) ;                           \
      FatalError(_error.str());                                        \
    }                                                                  \
} while(0)

bool advise;

void allocAdviseMemory(void** ppout,size_t outbuf_size,int gpuid) {
	checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors (hipMallocManaged (ppout, outbuf_size));
	if(advise)
	  checkCudaErrors (hipMemAdvise (*ppout,outbuf_size ,hipMemAdviseSetPreferredLocation,gpuid));
}

//seems to be a class like struct
class baseModule{
	public : 
	float alpha;
	float beta;

	string name;
  int in_channels, out_channels ;
  int in_width, in_height, out_width, out_height;

	float * pin; // this is pass in from outside
	float * pinDiff;
	long inbuf_size;
	float * pout; // pout should be alloc by child class
	float * poutDiff;
	long outbuf_size;

	int minibatch;

	int gpuid;

  hipdnnHandle_t cudnnHandle;
  hipblasHandle_t cublasHandle;

	size_t m_workspaceSizeByte;
	void * p_workspace;

	bool bNeedSyncInTensor;


	void print() {
					cout<<"name "<<name<<endl;
					cout<<"in_channels "<<in_channels<<endl;
					cout<<"out_channels "<<out_channels<<endl;
					cout<<"in_width "<<in_width<<endl;
					cout<<"in_height "<<in_height<<endl;
					cout<<"out_width "<<out_width<<endl;
					cout<<"out_height "<<out_height<<endl;
					cout<<"inbuf_size "<<inbuf_size<<endl;
					cout<<"outbuf_size "<<outbuf_size<<endl;
					cout<<"minibatch "<<minibatch<<endl;
					cout<<"gpuid "<<gpuid<<endl;
	}
	baseModule(
									float alpha_,
									float beta_,
									string name_,
  								hipdnnHandle_t cudnnHandle_,
								  hipblasHandle_t cublasHandle_,
									int gpuid_,
									int minibatch_,
									int in_c_,
									int out_c_,
									int in_h_,
									int in_w_,
									int out_h_,
									int out_w_,
									float * pin_
									) 
	{
		alpha=alpha_;
		beta=beta_;
		name=name_;
		cudnnHandle = cudnnHandle_;
		cublasHandle = cublasHandle_;
		gpuid = gpuid_;
		minibatch = minibatch_;
		in_channels = in_c_;
		out_channels = out_c_;
		in_width = in_w_;
		in_height = in_h_;
		out_width = out_w_;
		out_height = out_h_;
		pin = pin_;
		inbuf_size = minibatch_*in_c_*in_w_*in_h_;
		m_workspaceSizeByte=0;
		p_workspace=NULL;
		bNeedSyncInTensor=true;

		assert(gpuid>=0);
		assert(minibatch >0);
		assert(in_channels >0);
		assert(out_channels >0);
		assert(in_width >0);
		assert(in_height >0);
		assert(out_width >0);
		assert(out_height >0);
		assert(pin );
	}

	void allocPout(int minibatch_,int out_c_,int out_h_,int out_w_) {
//		minibatch = minibatch_;
//		out_channels = out_c_;
//		out_width = out_w_;
//		out_height = out_h_;
					assert(minibatch_==minibatch);
					assert(out_channels==out_c_);
					assert(out_width==out_w_);
					assert(out_height==out_h_);
		outbuf_size = minibatch_*out_c_*out_w_*out_h_;
		allocAdviseMemory((void**)&pout,sizeof (float) *outbuf_size,gpuid);
	}

	void allocPinDiff() {
		allocAdviseMemory((void**)&pinDiff,sizeof (float) *inbuf_size,gpuid);
	}

	virtual void fw1step() {};
	virtual void bw1step() {};

	~baseModule  () {
			checkCudaErrors(hipSetDevice(gpuid));
			hipFree(pout);
			hipFree(pinDiff);
	}
	size_t getOutputFloatNumber() {
					return outbuf_size;
	}
	size_t getInputFloatNumber() {
					return inbuf_size;
	}

};

class MaxPoolLayer: public baseModule {
	public :
	int size, stride;
	hipdnnTensorDescriptor_t srcTensorDesc;
  hipdnnPoolingDescriptor_t poolDesc;
	hipdnnTensorDescriptor_t  dstTensorDesc; //this out already have pout in baseModule
	MaxPoolLayer(
			float alpha_,
			float beta_,
			string name_,
			hipdnnHandle_t cudnnHandle_,
			hipblasHandle_t cublasHandle_,
			int gpuid_,
			int minibatch_,
			int in_channels_, 
			int in_h_, int in_w_, 
			int kernel_size_, int stride_, //it seems pooling just remain the same number of channel as input
			int paddingH_, int paddingW_,float * pin_) 
					: baseModule(
									alpha_,
									beta_,
									name_,
									cudnnHandle_,
									cublasHandle_,
									gpuid_,
									minibatch_,
									in_channels_,
									in_channels_, // it seems the output channle is the same as input
									in_h_,
									in_w_,
									(in_h_+paddingH_*2-kernel_size_)/stride_+1,
									(in_w_+paddingW_*2-kernel_size_)/stride_+1,
									pin_
								)
	{
				printf("MaxPoolLayer gpuid %d minibatch %d in_channels_ %d in_h_ %d in_w_ %d kernel_size_ %d stride_ %d paddingH_ %d paddingW_ %d\n",
						                      gpuid_ ,  minibatch_ ,  in_channels_ ,  in_h_ ,  in_w_ ,   kernel_size_ ,  stride_ ,  paddingH_ ,  paddingW_ );
		size= kernel_size_;
		stride = stride_;
		assert(size > 0);
		assert(stride > 0);
//		assert((in_w_+paddingW_*2-kernel_size_)%stride_ == 0);
//		assert((in_h_+paddingH_*2-kernel_size_)%stride_ == 0);

		allocPout(minibatch,in_channels_,(in_h_+paddingH_*2-kernel_size_)/stride_+1,(in_w_+paddingW_*2-kernel_size_)/stride_+1);
		allocPinDiff();
		//all layer follow this pattern
		// 1 set the source tensor
		// 2 set the operator tensor
		// 3 set the dest tensor

		// 1 set the source tensor
    checkCUDNN (hipdnnCreateTensorDescriptor (&srcTensorDesc));
    checkCUDNN(hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                              HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT,
																							minibatch, in_channels, in_height, in_width
                                              ));

		// 2 set the operator tensor
    checkCUDNN(hipdnnCreatePoolingDescriptor(&poolDesc));
    checkCUDNN(hipdnnSetPooling2dDescriptor(poolDesc,
                                               HIPDNN_POOLING_MAX,
                                               HIPDNN_PROPAGATE_NAN,
                                               size, size,
                                               paddingH_,paddingW_ ,
                                               stride, stride));

		// 3 set the dest tensor
    checkCUDNN (hipdnnCreateTensorDescriptor (&dstTensorDesc));
    checkCUDNN (hipdnnSetTensor4dDescriptor (dstTensorDesc,
					    HIPDNN_TENSOR_NCHW,
					    HIPDNN_DATA_FLOAT, minibatch_, in_channels_, (in_h_+paddingH_*2-kernel_size_+1)/stride_, (in_w_+paddingW_*2-kernel_size_+1)/stride_));
		


		// should not be sync
		bNeedSyncInTensor = false;
		m_workspaceSizeByte =0;

	}
	void fw1step () {
				//pooling layer dont need workspace
				//assert(p_workspace!=NULL);
				//assert(m_workspaceSizeByte!=0);
        checkCudaErrors(hipSetDevice(gpuid));
        checkCUDNN(hipdnnPoolingForward(cudnnHandle, poolDesc, &alpha, srcTensorDesc,
                                       pin, &beta, dstTensorDesc,pout));

	}
	void bw1step() {
					// pooling layer have no weight
    checkCudaErrors(hipSetDevice(gpuid));
		checkCUDNN(hipdnnPoolingBackward(cudnnHandle, poolDesc, &alpha, 
                                        dstTensorDesc, pout,dstTensorDesc, poutDiff,
                                        srcTensorDesc, pin, &beta, srcTensorDesc, pinDiff));
	}
	~MaxPoolLayer() {
		checkCUDNN (hipdnnDestroyTensorDescriptor(srcTensorDesc));
		checkCUDNN (hipdnnDestroyPoolingDescriptor(poolDesc));
    checkCUDNN (hipdnnDestroyTensorDescriptor (dstTensorDesc));
	}
};

class ConvBiasLayer: public baseModule
{
		public :
		int kernel_size,stride;

		hipdnnTensorDescriptor_t biasTensor;
		float * pconvbias;
		float * pconvbiasGradient;

		hipdnnTensorDescriptor_t srcTensorDesc;

		hipdnnFilterDescriptor_t filterDesc;
		float * pconvWeigth;
		float * pconvWeigthGradient;

		hipdnnConvolutionDescriptor_t convDesc;

		hipdnnTensorDescriptor_t  dstTensorDesc; //this out already have pout in baseModule

		hipdnnConvolutionFwdAlgo_t fwalgo;
		hipdnnConvolutionBwdFilterAlgo_t bwfalgo;
		hipdnnConvolutionBwdDataAlgo_t bwdalgo;

    ConvBiasLayer (
				float alpha_,
				float beta_,
				string name_,
				hipdnnHandle_t cudnnHandle_,
			  hipblasHandle_t cublasHandle_,
				int gpuid_,
				int minibatch_,
				int in_channels_, 
				int in_h_, int in_w_, 
				int numFilter_, int kernel_size_, int stride_, 
				int paddingH_, int paddingW_,
				float * pin_)  // pin pass from outside
						: baseModule(
									alpha_,
									beta_,
									name_,
									cudnnHandle_,
									cublasHandle_,
									gpuid_,
									minibatch_,
									in_channels_,
									numFilter_,
									in_h_,
									in_w_,
									(in_h_+paddingH_*2-kernel_size_)/stride_+1,
									(in_w_+paddingW_*2-kernel_size_)/stride_+1,
									pin_
								)
		{
				printf("ConvBiasLayer gpuid %d minibatch %d in_channels_ %d in_h_ %d in_w_ %d numFilter_ %d kernel_size_ %d stride_ %d paddingH_ %d paddingW_ %d out_height %d out_width %d\n",
						                      gpuid ,  minibatch ,  in_channels_ ,  in_h_ ,  in_w_ ,  numFilter_ ,  kernel_size_ ,  stride_ ,  paddingH_ ,  paddingW_ , (in_h_+paddingH_*2-kernel_size_)/stride_+1, (in_w_+paddingW_*2-kernel_size_)/stride_+1);
				//assert((in_w_+paddingW_*2-kernel_size_)%stride_ == 0);
				//assert((in_h_+paddingH_*2-kernel_size_)%stride_ == 0);

				kernel_size = kernel_size_;
				assert(kernel_size<16); //this is not strict, just to prevent unreasonable large kernel
				stride=stride_;
				assert(stride < 16);//also not strict

				//bias descriptor
				checkCUDNN (hipdnnCreateTensorDescriptor (&biasTensor));
    		checkCUDNN (hipdnnSetTensor4dDescriptor (biasTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, out_channels, 1, 1));
    		allocAdviseMemory((void**)&pconvbias,         sizeof(float) * out_channels,gpuid );
    		allocAdviseMemory((void**)&pconvbiasGradient, sizeof(float) * out_channels,gpuid );

				//set the source tensor
    		checkCUDNN (hipdnnCreateTensorDescriptor (&srcTensorDesc));
				//this may fail in 800 pixel because the hipdnnSetTensor4dDescriptor require the tensor smaller than 2GB, so I may need 700
    		checkCUDNN (hipdnnSetTensor4dDescriptor (srcTensorDesc,
					    HIPDNN_TENSOR_NCHW,
					    HIPDNN_DATA_FLOAT, minibatch, in_channels, in_height, in_width));

				//set the filter desc
    		checkCUDNN (hipdnnCreateFilterDescriptor (&filterDesc));
    		checkCUDNN (hipdnnSetFilter4dDescriptor (filterDesc,
					    HIPDNN_DATA_FLOAT,
					    HIPDNN_TENSOR_NCHW,
					    out_channels,
					    in_channels,
					    kernel_size,
					    kernel_size));
				allocAdviseMemory((void**)&pconvWeigth,        sizeof(float)*in_channels_*kernel_size_*kernel_size_*numFilter_,gpuid);
				allocAdviseMemory((void**)&pconvWeigthGradient,sizeof(float)*in_channels_*kernel_size_*kernel_size_*numFilter_,gpuid);

    		checkCUDNN (hipdnnCreateConvolutionDescriptor (&convDesc));
    		checkCUDNN (hipdnnSetConvolution2dDescriptor (convDesc,
						 paddingH_, paddingW_,
						 stride, stride,
						 1, 1, // we currently dont support dilation
						 HIPDNN_CROSS_CORRELATION,
						 HIPDNN_DATA_FLOAT));

				int n,c,h,w;
    		checkCUDNN (hipdnnGetConvolution2dForwardOutputDim (convDesc,
						       srcTensorDesc,
						       filterDesc,
						       &n, &c, &h, &w));
				assert(n==minibatch);
				assert(c=out_channels);
				assert(h==out_height);
				assert(w==out_width);
				cout<<"minibatch "<<minibatch<<endl;
				cout<<"out_channels "<<out_channels<<endl;
				cout<<"out_height "<<out_height<<endl;
				cout<<"out_width "<<out_width<<endl;

				allocPout(n,c,h,w);
				allocPinDiff();

    		checkCUDNN (hipdnnCreateTensorDescriptor (&dstTensorDesc));
    		checkCUDNN (hipdnnSetTensor4dDescriptor (dstTensorDesc,
					    HIPDNN_TENSOR_NCHW,
					    HIPDNN_DATA_FLOAT, n, c, h, w));
		    checkCUDNN (hipdnnGetConvolutionForwardAlgorithm (cudnnHandle,
						     srcTensorDesc,
						     filterDesc,
						     convDesc,
						     dstTensorDesc,
						     HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
						     0, &fwalgo));

		    checkCUDNN (hipdnnGetConvolutionForwardWorkspaceSize (cudnnHandle,
							 srcTensorDesc,
							 filterDesc,
							 convDesc,
							 dstTensorDesc,
							 fwalgo, &m_workspaceSizeByte));
				//assert(m_workspaceSizeByte >0);

				bNeedSyncInTensor = kernel_size_ >1; // bigger than 1 need to consider data from neighbour

				//handling backward algo
        // If backprop filter algorithm was requested
				size_t tmpsize=0;
        checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(
            cudnnHandle, srcTensorDesc, dstTensorDesc, convDesc, filterDesc,
            HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 0, &bwfalgo));

        checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(
            cudnnHandle, srcTensorDesc, dstTensorDesc, convDesc, filterDesc, 
            bwfalgo, &tmpsize));

        m_workspaceSizeByte = std::max(m_workspaceSizeByte, tmpsize);

        // If backprop data algorithm was requested
        checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(
            cudnnHandle, filterDesc, dstTensorDesc, convDesc, srcTensorDesc,
            HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 0, &bwdalgo));

        checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(
            cudnnHandle, filterDesc, dstTensorDesc, convDesc, srcTensorDesc, 
            bwdalgo, &tmpsize));

        m_workspaceSizeByte = std::max(m_workspaceSizeByte, tmpsize);
        
		}
	void fw1step () {
					//this may not be neccessary
//				if(p_workspace==NULL) 
//					cout<<"WARNING : gpuid "<< gpuid<<" p_workspace is empty"<<endl;
//				if(m_workspaceSizeByte==0) 
//					cout<<"WARNING : gpuid "<< gpuid<<" m_workspaceSizeByte is zero"<<endl;
        float alpha = 1.0f, beta = 0.0f;
        checkCudaErrors(hipSetDevice(gpuid));
        checkCUDNN(hipdnnConvolutionForward(cudnnHandle, 
																					&alpha, 
																					srcTensorDesc, pin, 
																					filterDesc, pconvWeigth, 
																					convDesc, 
                                           fwalgo, p_workspace, m_workspaceSizeByte, &beta,
                                           dstTensorDesc, pout));
        checkCUDNN(hipdnnAddTensor(cudnnHandle, &alpha, biasTensor,
                                  pconvbias, &alpha, dstTensorDesc, pout));

	}

	void bw1step () {
        checkCUDNN(hipdnnConvolutionBackwardBias(cudnnHandle, &alpha, dstTensorDesc,
                                                poutDiff, &beta, biasTensor, pconvbias));

        
        checkCUDNN(hipdnnConvolutionBackwardFilter(cudnnHandle, &alpha, srcTensorDesc,
                                                  pin, dstTensorDesc, poutDiff, convDesc,
                                                  bwfalgo, p_workspace, m_workspaceSizeByte,
                                                  &beta, filterDesc, pconvWeigthGradient));
    
        checkCUDNN(hipdnnConvolutionBackwardData(cudnnHandle, &alpha, filterDesc,
                                                pconvWeigth, dstTensorDesc, poutDiff, convDesc, 
                                                bwdalgo, p_workspace, m_workspaceSizeByte,
                                                &beta, srcTensorDesc, pinDiff));
			
	}
	~ConvBiasLayer() {
		checkCUDNN (hipdnnDestroyTensorDescriptor(biasTensor));
		checkCUDNN (hipdnnDestroyTensorDescriptor(srcTensorDesc));
		checkCUDNN (hipdnnDestroyFilterDescriptor(filterDesc));
		checkCudaErrors(hipFree(pconvWeigth));
		checkCudaErrors(hipFree(pconvWeigthGradient));
		checkCudaErrors(hipFree(pconvbias));
		checkCudaErrors(hipFree(pconvbiasGradient));
		checkCUDNN (hipdnnDestroyConvolutionDescriptor(convDesc));
    checkCUDNN (hipdnnDestroyTensorDescriptor (dstTensorDesc));
	}

};


///////////////////////////////////////////////////////////////////////////////////////////
// CUDNN/CUBLAS training context

class TrainingContext
{
	public :
	float alpha,beta; 
  hipdnnHandle_t cudnnHandle;
  hipblasHandle_t cublasHandle;
  int m_gpuid;
  int m_batchSize;
	//only the first tensor is need to store here
  hipdnnTensorDescriptor_t dataTensor;

	vector<class baseModule *> vmod;
	int currentlayer;

	void * pworkspace;

	void print() {
					printf("TrainingContext m_gpuid %d m_batchSize %d\n",m_gpuid,m_batchSize);
					for(int i =0;i<vmod.size();i++) {
						vmod[i]->print();
					}
	}

  TrainingContext (int gpuid, int batch_size,float alpha_,float beta_)
  {
    m_batchSize = batch_size;
		m_gpuid =gpuid;
    printf ("gpuid %d batch_size %d\n", gpuid,batch_size);
		currentlayer=0;
		alpha=alpha_;
		beta=beta_;

    // Create CUBLAS and CUDNN handles
    checkCudaErrors (hipSetDevice (gpuid));
    /*checkCudaErrors (*/hipblasCreate (&cublasHandle)/*)*/;
    checkCUDNN (hipdnnCreate (&cudnnHandle));

    // Create tensor descriptors
    checkCUDNN (hipdnnCreateTensorDescriptor (&dataTensor));
  }

	void addMod(class baseModule * pmod) {
		vmod.push_back(pmod);
	}

	baseModule * getCurrentLayer() {
			assert(currentlayer >=0);
			if(currentlayer >= vmod.size()) {
					cout<<"currentlayer "<<currentlayer<<"vmod.size "<<vmod.size()<<endl;
					assert(0);
			}
			return vmod[currentlayer];
	}

	baseModule * getLastLayer() {
		return vmod[vmod.size()-1];
	}

	void reset() {
					currentlayer=0;
	}

	bool isForwardFinished() {
		 if(currentlayer>=vmod.size()) return true;
		 else return false;
	}
	bool isBackwardFinished() {
		 if(currentlayer==0) return true;
		 else return false;
	}

	void finishAddMod () {
		size_t maxsize=0;
		for(int i=0;i<vmod.size();i++) {
			maxsize = max(maxsize,vmod[i]->m_workspaceSizeByte);
//			cout<<"maxsize "<<vmod[i]->m_workspaceSizeByte <<endl;
		}
		//alloc new size
		if(maxsize>0) {
			allocAdviseMemory(&pworkspace,maxsize,m_gpuid);
		} else {
						maxsize = 0;
						pworkspace=NULL;
		}
		for(int i=0;i<vmod.size();i++) {
				vmod[i]->p_workspace = pworkspace;
				vmod[i]->m_workspaceSizeByte=maxsize;
		}
	}

   ~TrainingContext ()
  {
		for(int i=0;i<vmod.size();i++) {
			delete vmod[i];
		}
    checkCudaErrors (hipSetDevice (m_gpuid));

    checkCUDNN (hipdnnDestroyTensorDescriptor (dataTensor));
    /*checkCudaErrors (*/hipblasDestroy (cublasHandle)/*)*/;
    checkCUDNN (hipdnnDestroy (cudnnHandle));
  }

	 void ForwardPropagation1() {
		 if(currentlayer>=vmod.size()) {
 		  cout<<"finished at layer "<<currentlayer<<endl;
			assert(0);
		 } else {
				cout<<"layer "<<currentlayer<<endl;
        checkCudaErrors(hipSetDevice(m_gpuid));

        // Conv1 layer
				vmod[currentlayer]->fw1step();
				
				currentlayer++;
		 }
	 }
	 void BackwardPropagation() {
		currentlayer--;
		 if(currentlayer<0) {
 		  cout<<"finished at layer "<<currentlayer<<endl;
			assert(0);
		 } else {
				cout<<"layer "<<currentlayer<<endl;
        checkCudaErrors(hipSetDevice(m_gpuid));

        // Conv1 layer
				vmod[currentlayer]->bw1step();
		 }
	 }
};//end of TrainingContext

__global__ void
ssyinitfloat (float *p, size_t n)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (size_t idx = index; idx < n; idx += stride)
    {
      p[idx] = 0.0;
    }
}

struct runingConfig {
	size_t width, height;
	int iterations;
	int minib;
//	int chnl;
	bool copy;
	float fract;
	int num_gpus;
  vector < float *> * pd_dataV;
	vector <TrainingContext * > * pcontextV;
};

void construct_Lenet(struct runingConfig * prc ){
  float alpha = 1.0f, beta = 0.0f;
	size_t width = prc->width;
	size_t height = prc->height;
//	int iterations = prc->iterations;
	int minib = prc->minib;
	int chnl = 3;
//	bool copy = prc->copy;
//	float fract = prc->fract;
	int num_gpus = prc->num_gpus;
	vector < float *> * pd_dataV=prc->pd_dataV;
	vector <TrainingContext * > * pcontextV = prc->pcontextV;
  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
    {
      checkCudaErrors (hipSetDevice (gpuid));
			//alloc the input data
			float * pdata;
			size_t input_sz = minib*chnl*width*height;
			checkCudaErrors(hipMallocManaged(&pdata,sizeof(float)*input_sz));
			//the context for this gpu
      TrainingContext * pcontext = new TrainingContext (gpuid, minib,alpha,beta);

      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											alpha,
											beta,
											"conv1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											chnl,
											height,width, 
											20,5,1,
											0,0,
											pdata
											);
			class MaxPoolLayer * ppool1=new MaxPoolLayer (
											alpha,
											beta,
											"pool1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pconv1->out_channels,
											pconv1->out_height, pconv1->out_width,
											2,2,
											0,0,
											pconv1->pout
											);
			class ConvBiasLayer * pconv2=new ConvBiasLayer (
											alpha,
											beta,
											"conv2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											ppool1->out_channels,
											ppool1->out_height, ppool1 ->out_width,
											50,5,1,
											0,0,
											ppool1->pout
											);
			class MaxPoolLayer * ppool2=new MaxPoolLayer (
											alpha,
											beta,
											"pool2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pconv2->out_channels,
											pconv2->out_height, pconv2->out_width,
											2,2,
											0,0,
											pconv2->pout
											);

			pcontext -> addMod(pconv1);
			pcontext -> addMod(ppool1);
			pcontext -> addMod(pconv2);
			pcontext -> addMod(ppool2);
			pcontext -> finishAddMod();

      pcontextV->push_back (pcontext);
			pd_dataV->push_back(pdata);
	}

	for (int gpuid = 0; gpuid < num_gpus; gpuid++) {
					(*pcontextV)[gpuid]-> print();
	}
}
void construct_Resnet(struct runingConfig * prc ){
  float alpha = 1.0f, beta = 0.0f;
	size_t width = prc->width;
	size_t height = prc->height;
//	int iterations = prc->iterations;
	int minib = prc->minib;
	int chnl = 3;
//	bool copy = prc->copy;
//	float fract = prc->fract;
	int num_gpus = prc->num_gpus;
	vector < float *> * pd_dataV=prc->pd_dataV;
	vector <TrainingContext * > * pcontextV = prc->pcontextV;
  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
    {
      checkCudaErrors (hipSetDevice (gpuid));
			//alloc the input data
			float * pdata;
			size_t input_sz = minib*chnl*width*height;
			checkCudaErrors(hipMallocManaged(&pdata,sizeof(float)*input_sz));
			//the context for this gpu
      TrainingContext * pcontext = new TrainingContext (gpuid, minib,alpha,beta);

      class ConvBiasLayer * pconv11=new ConvBiasLayer (
											alpha,
											beta,
											"conv1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											chnl,
											height,width, 
											64,7,2, //output filters number, kernel size,stride
											0,0,//padding
											pdata
											);
			pcontext -> addMod(pconv11);
			class MaxPoolLayer * ppool1=new MaxPoolLayer (
											alpha,
											beta,
											"conv2_pool",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											3,2,//kernel size, stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
			pcontext -> addMod(ppool1);
			//conv2
			for(int i =0;i<3;i++) {
	      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											alpha,
											beta,
											"conv2_"+to_string(i)+"_1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											64,1,1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv1);

				class ConvBiasLayer * pconv2=new ConvBiasLayer (
											alpha,
											beta,
											"conv2_"+to_string(i)+"_2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											64,3,1,
											1,1,
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv2);
	      class ConvBiasLayer * pconv3=new ConvBiasLayer (
											alpha,
											beta,
											"conv2_"+to_string(i)+"_3",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											256,1,1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv3);
			}
			//conv3
			for(int i =0;i<4;i++) {
	      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											alpha,
											beta,
											"conv3_"+to_string(i)+"_1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											128,1,(i==0)?2:1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv1);

				class ConvBiasLayer * pconv2=new ConvBiasLayer (
											alpha,
											beta,
											"conv3_"+to_string(i)+"_2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											128,3,1,
											1,1,
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv2);
	      class ConvBiasLayer * pconv3=new ConvBiasLayer (
											alpha,
											beta,
											"conv3_"+to_string(i)+"_3",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											512,1,1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv3);
			}
			//conv4
			for(int i =0;i<23;i++) {
	      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											alpha,
											beta,
											"conv4_"+to_string(i)+"_1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											256,1,(i==0)?2:1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv1);

				class ConvBiasLayer * pconv2=new ConvBiasLayer (
											alpha,
											beta,
											"conv4_"+to_string(i)+"_2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											256,3,1,
											1,1,
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv2);
	      class ConvBiasLayer * pconv3=new ConvBiasLayer (
											alpha,
											beta,
											"conv4_"+to_string(i)+"_3",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											1024,1,1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv3);
			}
			//conv5
			for(int i =0;i<3;i++) {
	      class ConvBiasLayer * pconv1=new ConvBiasLayer (
											alpha,
											beta,
											"conv5_"+to_string(i)+"_1",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											512,1,(i==0)?2:1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv1);

				class ConvBiasLayer * pconv2=new ConvBiasLayer (
											alpha,
											beta,
											"conv5_"+to_string(i)+"_2",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											512,3,1,
											1,1,
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv2);
	      class ConvBiasLayer * pconv3=new ConvBiasLayer (
											alpha,
											beta,
											"conv5_"+to_string(i)+"_3",
											pcontext->cudnnHandle,
											pcontext->cublasHandle,
											gpuid,
											minib,
											pcontext->getLastLayer()->out_channels,
											pcontext->getLastLayer()->out_height, pcontext->getLastLayer()->out_width,
											2048,1,1, //output filters number, kernel size,stride
											0,0,//padding
											pcontext->getLastLayer()->pout
											);
				pcontext -> addMod(pconv3);
			}

			pcontext -> finishAddMod();

			//setting the diff buffer pointer
			int lastl=(pcontext->vmod).size();
			size_t lastSizeInFloat = (pcontext->vmod)[lastl-1]->getOutputFloatNumber();
			float * plast;
    	checkCudaErrors (hipMallocManaged (&plast, sizeof (float) * lastSizeInFloat));
			for(int i=(pcontext->vmod).size()-1;i>=0;i--) {
				(pcontext->vmod)[i]->poutDiff=plast;
				plast=(pcontext->vmod)[i]->pinDiff;
			}

      pcontextV->push_back (pcontext);
			pd_dataV->push_back(pdata);
	}

	for (int gpuid = 0; gpuid < num_gpus; gpuid++) {
					(*pcontextV)[gpuid]-> print();
	}
}

void syncAllGPU(int num_gpus) {
  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
  {
    checkCudaErrors (hipSetDevice (gpuid));
    checkCudaErrors (hipDeviceSynchronize ());
  }
}

///////////////////////////////////////////////////////////////////////////////////////////
// Main function
//#define WIDTH 280
int
main (int argc, char **argv)
{
  if (argc != 8) {
		printf("Usage : cudnnModelParallel.exe <nettype> <iteration> <minbatch> <width>  <copy or not> <fract to copy> <advise or not>");
		assert(0);
	}

	cout<<"argc "<<argc<<endl;
	char * nettype =argv[1];
  int iterations = atoi (argv[2]);
	int minib = atoi(argv[3]);

  size_t width, height;
  width = atoi (argv[4]);
  height = width;
	cout<<"width "<<width<<endl;

  bool copy = (atoi (argv[5]) > 0);
  float fract = (atof (argv[6]));

  // Choose GPU
  int num_gpus;
  checkCudaErrors (hipGetDeviceCount (&num_gpus));
	cout<<"num_gpus "<<num_gpus<<endl;

	int deviceId;
//  int numberOfSMs;
	checkCudaErrors(hipSetDevice(0));
	hipGetDevice(&deviceId);
	cout<<"deviceId "<<deviceId<<endl;

	advise= (atoi(argv[7]) >0);
//	hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	//printf("numberOfSMs %s\n",numberOfSMs);
//  int threadsPerBlock = 256;
//  int numberOfBlocks = 32*80 ;


  vector < float *>d_dataV;
	vector <TrainingContext * >contextV;
	
	//construct the resnet
	struct runingConfig rc;
	rc.width =width;
	rc.height = height;
	rc.iterations = iterations;
	rc.minib = minib;
//	rc.chnl = chnl;
	rc.copy = copy;
	rc.fract = fract;
	rc.num_gpus=num_gpus;
	rc.pd_dataV = & d_dataV;
	rc.pcontextV= & contextV;

	if(strcmp(nettype,"resnet")==0) {
		cout<<"resnet"<<endl;
		construct_Resnet(&rc);
	} else if(strcmp(nettype,"lenet")==0) {
		cout<<"lenet"<<endl;
		construct_Lenet(&rc);
	}

	syncAllGPU(num_gpus);
  // Use SGD to train the network
	size_t totalsize=0;
  auto t1 = chrono::high_resolution_clock::now ();
  for (int iter = 0; iter < iterations; ++iter)
  {
		//reset
		for(int gpuid=0;gpuid<num_gpus;gpuid++)     {
						contextV[gpuid]->reset();
		}
		//forward propage
		while(true) {
			//run one layer
		  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			{
				assert(contextV[gpuid]->m_gpuid == gpuid);
			  checkCudaErrors (hipSetDevice (gpuid));
			  contextV[gpuid]->ForwardPropagation1 ();
				assert(contextV[gpuid]->currentlayer >0);
			}
			if(contextV[0]->isForwardFinished()) break;

			syncAllGPU(num_gpus);	

		  if (copy)
			{
			  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			  {
			      //sync n+1 to n
			      checkCudaErrors (hipSetDevice (gpuid));
						baseModule * pcurrent =contextV[gpuid]->getCurrentLayer();
			      size_t sz = sizeof (float) * (pcurrent->getInputFloatNumber() );
						assert(sz>0);
						cout<<"sz "<<sz<<endl;

			      if (gpuid > 0 && pcurrent-> bNeedSyncInTensor) {
							baseModule * pPrev =contextV[gpuid-1]->getCurrentLayer();
							size_t szPrev = sizeof (float) * (pPrev->getInputFloatNumber() );
							assert(sz==szPrev);
							size_t tobetransfered = int (fract * sz / 2);
							if(gpuid == num_gpus-1)
								totalsize = totalsize+tobetransfered;
						  checkCudaErrors (hipMemcpyAsync (pcurrent->pin + sz / (2 * sizeof (float)), pPrev->pin, tobetransfered, hipMemcpyDefault));
						} else {
							cout <<"No need to sync : gpuid "<<gpuid << "layer "<<pcurrent->name<<endl;
						}
			  }
		
				syncAllGPU(num_gpus);
			}
		}
		//backward propage
		while(true) {
			//run one layer
		  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			{
				assert(contextV[gpuid]->m_gpuid == gpuid);
			  checkCudaErrors (hipSetDevice (gpuid));
			  contextV[gpuid]->BackwardPropagation();
				assert(contextV[gpuid]->currentlayer >=0);
			}
			if(contextV[0]->isBackwardFinished()) break;

			syncAllGPU(num_gpus);	

		  if (copy)
			{
			  for (int gpuid = 0; gpuid < num_gpus; gpuid++)
			  {
			      //sync n+1 to n
			      checkCudaErrors (hipSetDevice (gpuid));
						baseModule * pcurrent =contextV[gpuid]->getCurrentLayer();
			      size_t sz = sizeof (float) * (pcurrent->getOutputFloatNumber() );
						assert(sz>0);
						cout<<"sz "<<sz<<endl;

			      if (gpuid > 0 && pcurrent-> bNeedSyncInTensor) {
							baseModule * pPrev =contextV[gpuid-1]->getCurrentLayer();
							size_t szPrev = sizeof (float) * (pPrev->getOutputFloatNumber() );
							assert(sz==szPrev);
							size_t tobetransfered = int (fract * sz / 2);
							if(gpuid == num_gpus-1)
								totalsize = totalsize+tobetransfered;
						  checkCudaErrors (hipMemcpyAsync (pcurrent->poutDiff + sz / (2 * sizeof (float)), pPrev->poutDiff, tobetransfered, hipMemcpyDefault));
						} else {
							cout <<"No need to sync : gpuid "<<gpuid << "layer "<<pcurrent->name<<endl;
						}
			  }
		
				syncAllGPU(num_gpus);
			}
		}
  }				// end of iteration
	syncAllGPU(num_gpus);
  auto t2 = chrono::high_resolution_clock::now ();

  cout<<"Iteration time: "
			<<" num_gpus "<< num_gpus
			<<" batch_size "<< minib
			<<" width "<<width
			<<" fract "<< (copy?fract:0.0)
			<<" totalsize "<< totalsize/iterations
			<<" advise "<< (advise?1:0)
			<<" time " << chrono::duration_cast < chrono::microseconds > (t2 - t1).count () / 1000.0f / iterations
		<<" ms"<<endl;
  return 0;
}
